#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "util.h"

#define THREADS_PER_BLOCK 256

__device__ int criteriaMetCounter = 0;

/**
 * @brief Checks for any error in CUDA operations.
 *        prints error message and stops the program for any CUDA error occured.
 * @param err hipError_t struct
 * @param lineNum interger represent the line number on which the error occured. used for error finding.
**/
void checkError(hipError_t err, int lineNum)
{
  if (err != hipSuccess)
  {
    fprintf(stderr, "CUDA Error on line %d: %s\n", lineNum, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

/**
 * @brief Calculates the distance between two points p1 and p2.
 * @param p1 Point 1
 * @param p2 Point 2
 * @return The distance between p1 and p2. -1 if p1 == p2.
**/
__device__ double calculateDistanceBetweenPoints(Point* p1, Point* p2)
{
  if (p1->id == p2->id)
    return -1;
  
  double xDist = pow((p2->x - p1->x), 2);
  double yDist = pow(p2->y - p1->y, 2);
  return sqrt(xDist + yDist);
}

/**
 * @brief Determines if the referenced point satisfies the proximity criteria.
 * @param ref The referenced Point.
 * @param points Array of all points.
 * @param size Size of points array
 * @param maximumDistance Maximum distance allowed between the reference point to other points to satisfiy the criteria.
 * @param minimumPoints Minimum points wihin the distance threshold required around the reference point to satisfiy the criteria.
 * @return 1 if the reference point satisfies the criteria, otherwise 0.
**/
__device__ int isPointSatisfiesCriteria(Point* ref, Point* points, int size, double maximumDistance, int minimumPoints)
{
  int count = 0;
  for (int i = 0; i < size && count < minimumPoints; i++)
  {
    double dist = calculateDistanceBetweenPoints(ref, &points[i]);
    count += dist >= 0 && dist < maximumDistance; // negative distance indicates distance to self
  }
  return count >= minimumPoints;
}

/**
 * @brief Helper device function to atomically reset the global criteria met counter.
**/
__device__ void resetCriteriaMetCounter()
{
  atomicExch(&criteriaMetCounter, 0);
}

/**
 * @brief Helper function to set the isCritetiraMet paramter to a referenced result struct.
 * @param res Refernce to the current result struct.
**/
__global__ void setResultMetadata(criteria_t* res)
{
  res->isCritetiraMet = criteriaMetCounter >= MIN_CRITERIA_POINTS;
  resetCriteriaMetCounter();
}

/**
 * @brief Calculates and sets the points positions for given time t.
 * @param points Array of points
 * @param size Size of points array
 * @param t Current time value
**/
__global__ void setPointsPositions(Point *points, int size, double t)
{
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadId > size)
    return;

  Point* p = &points[threadId];
  p->x = ((p->x2 - p->x1) / 2) * sin(t * M_PI_2) + ((p->x2 + p->x1) / 2);
  p->y = p->a * p->x + p->b;
}

/**
 * @brief Checks which points satisfies the proximity criteria in a given time t. results are saved in the referenced result struct.
 * @param points Array of points
 * @param size Size of points array
 * @param maximumDistance Maximum distance allowed between the reference point to other points to satisfiy the criteria.
 * @param minimumPoints Minimum points wihin the distance threshold required around the reference point to satisfiy the criteria.
 * @param result Refernce to the current result struct
**/
__global__ void checkProximityCriteria(Point *points, int size, double maximumDistance, int minimumPoints, criteria_t* result)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid > size|| atomicAdd(&criteriaMetCounter, 0)  >= MIN_CRITERIA_POINTS)
    return;
  
  if (isPointSatisfiesCriteria(&points[tid], points, size, maximumDistance, minimumPoints))
  {
    /* 
      try to claim the right to increment criteriaMetCounter and update the results array.
      this is effectively a spin-lock for the right to update the results array.
    */
    while (atomicAdd(&criteriaMetCounter, 0) < MIN_CRITERIA_POINTS)
    {
      int expectedValue = criteriaMetCounter;
      int newValue = expectedValue + 1;
      int value = atomicCAS(&criteriaMetCounter, expectedValue, newValue);

      if (value == expectedValue && value < MIN_CRITERIA_POINTS)
      {
        result->pointIDs[value] = points[tid].id;
        break;
      }
    }
  }
}

void computeProximities(Point *h_points, int size, criteria_t *h_results, int chunk, double maximumDistance, int minimumPoints)
{
  hipError_t err;
  int i, blocks;

  // allocate device memory
  Point *d_points;
  err = hipMalloc((void **)&d_points, size * sizeof(Point));
  checkError(err, __LINE__ - 1);

  criteria_t *d_results;
  err = hipMalloc((void **)&d_results, chunk * sizeof(criteria_t));
  checkError(err, __LINE__ - 1);

  // copy data to device
  err = hipMemcpy(d_points, h_points, size * sizeof(Point), hipMemcpyHostToDevice);
  checkError(err, __LINE__ - 1);

  err = hipMemcpy(d_results, h_results, chunk * sizeof(criteria_t), hipMemcpyHostToDevice);
  checkError(err, __LINE__ - 1);
  
  blocks = (int)floor((size * chunk) / THREADS_PER_BLOCK) + 1; // +1 handles case where size * chunk < THREADS_PER_BLOCK

  for (i = 0; i < chunk; i++)
  {
    setPointsPositions<<<blocks, THREADS_PER_BLOCK>>>(d_points, size, h_results[i].t);
    checkError(hipGetLastError(), __LINE__ - 1);
    
    checkProximityCriteria<<<blocks, THREADS_PER_BLOCK>>>(d_points, size, maximumDistance, minimumPoints, &d_results[i]);
    checkError(hipGetLastError(), __LINE__ - 1);
    
    setResultMetadata<<<1, 1>>>(&d_results[i]);
    checkError(hipGetLastError(), __LINE__ - 1);
    hipDeviceSynchronize(); // wait for all threads to finish with current timestamp
  }
  // copy results to host
  err = hipMemcpy(h_results, d_results, chunk * sizeof(criteria_t), hipMemcpyDeviceToHost);
  checkError(err, __LINE__ - 1);

  // free allocated device memory
  hipFree(d_points);
  hipFree(d_results);
}
