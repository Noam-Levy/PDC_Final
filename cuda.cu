#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "util.h"

#define THREADS_PER_BLOCK 256

__device__ int criteriaMetCounter = 0;

void checkError(hipError_t err, int lineNum)
{
  if (err != hipSuccess)
  {
    fprintf(stderr, "CUDA Error on line %d: %s\n", lineNum, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

__device__ double calculateDistanceBetweenPoints(Point* p1, Point* p2)
{
  if (p1->id == p2->id)
    return -1;
  
  float xDist = pow((p2->x - p1->x), 2);
  float yDist = pow(p2->y - p1->y, 2);
  return sqrt(xDist + yDist);
}

__device__ int isPointSatisfiesCriteria(Point* ref, Point* points, int size, double minimumDistance, int minimumPoints)
{
  int count = 0;
  for (int i = 0; i < size; i++)
  {
    if (points[i].id != ref->id)
    {
      double dist = calculateDistanceBetweenPoints(ref, &points[i]);
      count += dist >= 0 && dist < minimumDistance; // negative distance indicates distance to self
    }    
  }
  return count >= minimumPoints;
}

__device__ void resetCriteriaMetCounter()
{
  criteriaMetCounter = 0;
}

__global__ void setResultMetadata(criteria_t* res, double t)
{
  res->t = t;
  res->isFound = criteriaMetCounter >= MIN_CRITERIA_POINTS;
  resetCriteriaMetCounter();
}

__global__ void setPointsPositions(Point *points, int size, double t)
{
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadId > size)
    return;

  Point p = points[threadId];
  p.x = ((p.x2 - p.x1) / 2) * sin(t * M_PI_2) + ((p.x2 + p.x1) / 2);
  p.y = p.a * p.x + p.b;
}

__global__ void checkProximityCriteria(Point *points, int size, double t, double minimumDistance, int minimumPoints, criteria_t* result)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid > size || atomicAdd(&criteriaMetCounter, 0)  >= MIN_CRITERIA_POINTS)
    return;
  
  if (isPointSatisfiesCriteria(&points[tid], points, size, minimumDistance, minimumPoints))
  {
    int counter = atomicAdd(&criteriaMetCounter, 1); // counter recieves the previous value of criteriaMetCounter.
    if (counter < MIN_CRITERIA_POINTS)
      result->pointIDs[counter] = points[tid].id;
  }
}

void computeProximities(Point *h_points, int size, double* h_times, criteria_t *h_results, int chunk, double minimumDistance, int minimumPoints)
{
  hipError_t err;
  int i, blocks;

  // allocate device memory
  Point *d_points;
  err = hipMalloc((void **)&d_points, size * sizeof(Point));
  checkError(err, __LINE__ - 1);

  criteria_t *d_results;
  err = hipMalloc((void **)&d_results, chunk * sizeof(criteria_t));
  checkError(err, __LINE__ - 1);

  // copy data to device
  err = hipMemcpy(d_points, h_points, size * sizeof(Point), hipMemcpyHostToDevice);
  checkError(err, __LINE__ - 1);
  
  blocks = (int)floor((size * chunk) / THREADS_PER_BLOCK) + 1; // +1 handles case where size * chunk < THREADS_PER_BLOCK

  for (i = 0; i < chunk; i++)
  {
    double t = h_times[i];
    
    setPointsPositions<<<blocks, THREADS_PER_BLOCK>>>(d_points, size, t);
    checkError(hipGetLastError(), __LINE__ - 1);
    
    checkProximityCriteria<<<blocks, THREADS_PER_BLOCK>>>(d_points, size, t, minimumDistance, minimumPoints, &d_results[i]);
    checkError(hipGetLastError(), __LINE__ - 1);
    
    setResultMetadata<<<1, 1>>>(&d_results[i], t);
    checkError(hipGetLastError(), __LINE__ - 1);
    hipDeviceSynchronize(); // wait for all threads to finish with current timestamp
  }
  // copy results to host
  err = hipMemcpy(h_results, d_results, chunk * sizeof(criteria_t), hipMemcpyDeviceToHost);
  checkError(err, __LINE__ - 1);

  // free allocated device memory
  hipFree(d_points);
  hipFree(d_results);
}
