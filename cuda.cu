#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "util.h"

#define THREADS_PER_BLOCK 256

__device__ int lock = 0;
__device__ int criteriaMetCounter = 0;

void checkError(hipError_t err, int lineNum)
{
  if (err != hipSuccess)
  {
    fprintf(stderr, "CUDA Error on line %d: %s\n", lineNum, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

__device__ void aquireLock()
{
  while (atomicCAS(&lock, 0, 1) != 0);
}

__device__ void releaseLock()
{
  atomicExch(&lock, 0);
}

__device__ double calculateDistanceBetweenPoints(Point* p1, Point* p2)
{
  if (p1->id == p2->id)
    return -1;
  
  float xDist = pow((p2->x - p1->x), 2);
  float yDist = pow(p2->y - p1->y, 2);
  return sqrt(xDist + yDist);
}


__device__ int isPointSatisfiesCriteria(Point* ref, Point* points, int size, double minimumDistance, int minimumPoints)
{
  int count = 0;
  for (int i = 0; i < size; i++)
  {
    if (points[i].id != ref->id)
    {
      double dist = calculateDistanceBetweenPoints(ref, &points[i]);
      count += dist >= 0 && dist < minimumDistance; // negative distance indicates distance to self
    }    
  }
  return count >= minimumPoints;
}

__global__ void setCriteriaMetCounter(int value)
{
  criteriaMetCounter = value;
}

__global__ void setPointsPositions(Point *points, int size, double t)
{
  int threadId = blockIdx.x * blockDim.x + threadIdx.x;
  if (threadId > size)
    return;

  Point p = points[threadId];
  p.x = ((p.x2 - p.x1) / 2) * sin(t * M_PI_2) + ((p.x2 + p.x1) / 2);
  p.y = p.a * p.x + p.b;
}

__global__ void checkProximityCriteria(Point *points, int size, double t, double minimumDistance, int minimumPoints, criteria_t* result)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid > size)
    return;
  
  if (isPointSatisfiesCriteria(&points[tid], points, size, minimumDistance, minimumPoints))
  {
    // aquireLock();
    if(criteriaMetCounter < MIN_CRITERIA_POINTS)
    {
      result->pointIDs[criteriaMetCounter++] = points[tid].id;
      printf("%d satisfies criteria at %.2f. counter=%d\n", points[tid].id, t, criteriaMetCounter);
    }
    // releaseLock();
  }

  if (criteriaMetCounter >= MIN_CRITERIA_POINTS)
  {
    // aquireLock();
    if (!result->isFound)
    {
      result->isFound = 1;
      result->t = t;
    }
    // releaseLock();
  }
}

__global__ void printPoints(Point* points, int size)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i > size)
    return;
  
  printf("P[%d]: x=%.2f y=%.2f\n", points[i].id, points[i].x, points[i].y);
}

__global__ void printResults(criteria_t *results, int size)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i > size)
    return;

  criteria_t res = results[i];
  if (res.isFound)
  {
    printf("Points ");
    for (int j = 0; j < MIN_CRITERIA_POINTS - 1; j++)
      printf("%d, ", res.pointIDs[j]);
    printf("%d satisfy Proximity Criteria at t=%.2lf\n", res.pointIDs[MIN_CRITERIA_POINTS - 1], res.t);
  }
  else
  {
    printf("No joy at t=%.2f\n", res.t);
  }
}

void computeProximities(Point *h_points, int size, double* h_times, criteria_t *h_results, int chunk, int tCount, double minimumDistance, int minimumPoints)
{
  hipError_t err;
  int i, offset;

  // allocate device memory
  Point *d_points;
  err = hipMalloc((void **)&d_points, size * sizeof(Point));
  checkError(err, __LINE__ - 1);

  criteria_t *d_results;
  err = hipMalloc((void **)&d_results, chunk * sizeof(criteria_t));
  checkError(err, __LINE__ - 1);

  // copy data to device
  err = hipMemcpy(d_points, h_points, size * sizeof(Point), hipMemcpyHostToDevice);
  checkError(err, __LINE__ - 1);
  
  int blocks = (int)floor((size * chunk) / THREADS_PER_BLOCK) + 1; // +1 handles case where size * chunk < THREADS_PER_BLOCK
  for (i = 0; i < chunk; i++)
  {
    setCriteriaMetCounter<<<1, 1>>>(0);
    double t = h_times[i];
    
    setPointsPositions<<<blocks, THREADS_PER_BLOCK>>>(d_points, size, t);
    checkError(hipGetLastError(), __LINE__ - 1);
    
    checkProximityCriteria<<<blocks, THREADS_PER_BLOCK>>>(d_points, size, t, minimumDistance, minimumPoints, &d_results[i]);
    checkError(hipGetLastError(), __LINE__ - 1);
  }
  hipDeviceSynchronize();

  // copy results to host
  err = hipMemcpy(h_results, d_results, chunk * sizeof(criteria_t), hipMemcpyDeviceToHost);
  checkError(err, __LINE__ - 1);

  // free allocated device memory
  hipFree(d_points);
  hipFree(d_results);
}
